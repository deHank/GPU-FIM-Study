#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <unordered_map>
#include <iostream>
#include <string.h>
#include <time.h>

#define MAX_NODES 6000  // Maximum nodes in the FP-Tree
#define EMPTY -1

typedef struct {
    int id; 
    int processed; // 1 signifies it was processed
    int itemSet;
    int count; 
    int parent;
    int nextSibling; 
    int firstChild; 
} Node; 

// Calculates the distance between two instances
__device__ float generateItemSet(float* instance_A, float* instance_B, int num_attributes) {
    float sum = 0;
    
    for (int i = 0; i < num_attributes-1; i++) {
        float diff = instance_A[i] - instance_B[i];
        //printf("instance a and b were %.3f %.3f\n", instance_A[i] ,instance_B[i]);
        sum += diff*diff;
    }
    //printf("sum was %.3f\n,", sum);
    return sqrt(sum);
}

__global__ void processItemSets(char *inData, int minimumSetNum, int *d_Offsets, int totalRecords, int blocksPerGrid) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Shared memory is treated as a single contiguous block
    extern __shared__ int sharedMemory[];

    char* line = inData + d_Offsets[tid];
    bool inNumber = false;
    int itemCount = 0;
    int number = 0;
    int items[32];
    // Initialize the shared memory (done by thread 0 in each block)
    if (tid <= 9) {
        printf("are are in tid %d\n", tid);
        //Extract items from the input line
        for (char* current = line; *current != '\n' && *current != '\0'; current++) {
            if (*current >= '0' && *current <= '9') {
                number = number * 10 + (*current - '0');
                inNumber = true;
            } else if (inNumber) {
                
                items[itemCount] = number;
                itemCount++;
                number = 0; 
                inNumber = false;
                
            }
           
        }

        if (inNumber) {
             items[itemCount++] = number;
        }
        for(int i = 0; i < itemCount; i++){
            printf("%d", items[i]);
            
        }
        
        
    }
    __syncthreads();

    // Parse the input and build the FP-Tree
    if (tid < totalRecords) {
        
    }


    
}

// Implements a threaded kNN where for each candidate query an in-place priority queue is maintained to identify the nearest neighbors
int KNN() {   
    clock_t cpu_start_withSetup = clock();
    
    clock_t setupTimeStart = clock();
    //int lineCountInDataset = 1692081;
    int lineCountInDataset = 55012;
    const char* inDataFilePath = "../sortedDataBase.txt";

    FILE* file = fopen(inDataFilePath, "r");

    // Get the file size
    fseek(file, 0, SEEK_END);
    size_t file_size = ftell(file);
    rewind(file);

    char* h_buffer = (char*)malloc(file_size);
    fread(h_buffer, 1, file_size, file);
    

    // Count the number of lines and create offsets
    int* h_offsets = (int*)malloc((file_size + 1) * sizeof(int));
    int lineCount = 0;
    h_offsets[lineCount++] = 0; // First line starts at the beginning
    
    for (size_t i = 0; i < file_size; i++) {
        //printf("are we in size?");
        if (h_buffer[i] == '\n') {
            //printf("we are in the newline stuff");
            h_offsets[lineCount++] = i + 1; // Next line starts after '\n'
            
        }
    }
    
    // Allocate memory to hold the file contents
    char* h_text = (char*)malloc(file_size);

    // Read the file into the host buffer
    fread(h_text, 1, file_size, file);
    //fclose(file);
    //size_t sharedMemSize = (6 * MAX_NODES) * sizeof(int) +  1 * sizeof(int) ;  // 5 arrays + nodeCounter
    
    // Allocate memory on the GPU
    char* d_text;
    int* d_offsets; 
    hipMalloc(&d_text, file_size);
    hipMalloc(&d_offsets, lineCountInDataset * sizeof(int));

    // Copy the file contents to the GPU
    hipMemcpy(d_text, h_buffer, file_size, hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, lineCountInDataset * sizeof(int), hipMemcpyHostToDevice);
    int threadsPerBlock = 32;
    int blocksPerGrid = ((lineCountInDataset + threadsPerBlock) - 1) /  threadsPerBlock; //how do we know how many blocks we need to use?
    //printf("BlocksPerGrid = %d\n", blocksPerGrid);
    printf("number of threads is roughly %d\n", threadsPerBlock*blocksPerGrid);


    

    int minItemCount = 3; //setting the minimum # of items to be considered an itemset

    //here I would want to generate all itemsets

    clock_t setupTimeEnd = clock();

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float cudaElapsedTime;

    
    hipEventRecord(startEvent);
    processItemSets<<<blocksPerGrid, threadsPerBlock>>>(d_text, minItemCount, d_offsets, lineCountInDataset, blocksPerGrid);
    hipDeviceSynchronize();
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);

    // Print the elapsed time (milliseconds)
    hipEventElapsedTime(&cudaElapsedTime, startEvent, stopEvent);
    printf("CUDA Kernel Execution Time: %.3f ms\n", cudaElapsedTime);

    // ensure there are no kernel errors
    hipError_t hipError_t = hipGetLastError();
    if(hipError_t != hipSuccess) {
        fprintf(stderr, "processItemSets cudaGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

    clock_t retrieveGPUResultsStart = clock();
    clock_t retrieveGPUResultsEnd = clock();

    // global reduction will be written to file
    FILE *resultsFile = fopen("cudaItemSetMiningResults.txt", "w");
    if (resultsFile == NULL) {
        perror("Error opening results file");
        return 1;
    }
    

    // Record end time
    clock_t cpu_end_withSetup = clock();
    // Calculate elapsed time in milliseconds
    // float cpuElapsedTime = ((float)(cpu_end - cpu_start)) / CLOCKS_PER_SEC * 1000.0;
    // float cpuElapsedTimeSetup = ((float)(cpu_end_withSetup - cpu_start_withSetup)) / CLOCKS_PER_SEC * 1000.0;
    // float setupTime = ((float)(setupTimeEnd - setupTimeStart)) / CLOCKS_PER_SEC * 1000.0;
    // float gpuRetrievalTime = ((float)(retrieveGPUResultsEnd - retrieveGPUResultsStart)) / CLOCKS_PER_SEC * 1000.0;

    // printf("CPU Execution Time: %.3f ms\n", cpuElapsedTime);
    // printf("Total Runtime: %.3f ms\n", cudaElapsedTime + cpuElapsedTime);
    // printf("Total Runtime (with setup/file write): %.3f ms\n", cpuElapsedTimeSetup);
    // printf("Total Setup Time: %.3f ms\n", setupTime);
    // printf("Total GPU Results Retrieval Time: %.3f ms\n", gpuRetrievalTime);
    //printf("Proccessed %d nodes\n", totalNodes);
    // // Print the aggregated counts (if has no child then follow up to the parent)
    // printf("{ ");
    // for (const auto& [itemSet, count] : map) {
    //     std::cout << itemSet << ": " << count << '\n';
    // } printf("}");
    return 1;
}

int main(int argc, char *argv[])
{
    

    int x = KNN();
    return -1;  
}